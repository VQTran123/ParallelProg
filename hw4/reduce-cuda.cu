#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <stdbool.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#define warpSize 32
#define blockSize 1024
#define nIsPow2 true

__device__ __forceinline__ double warpReduceSum(unsigned int mask, double mySum) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    mySum += __shfl_down_sync(mask, mySum, offset);
  }
  return mySum;
}

__global__ void reduce7(double *g_idata, double *g_odata,
                        unsigned int n) {
  extern __shared__ double __smem_d[];
  double *sdata = (double *)__smem_d;

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int gridSize = blockSize * gridDim.x;
  unsigned int maskLength = (blockSize & 31);  // 31 = warpSize-1
  maskLength = (maskLength > 0) ? (32 - maskLength) : maskLength;
  const unsigned int mask = (0xffffffff) >> maskLength;

  int mySum = 0;

  // we reduce multiple num per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer num per thread
  if (nIsPow2) {
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    gridSize = gridSize << 1;

    while (i < n) {
      mySum += g_idata[i];
      // ensure we don't read out of bounds -- this is optimized away for
      // powerOf2 sized arrays
      if ((i + blockSize) < n) {
        mySum += g_idata[i + blockSize];
      }
      i += gridSize;
    }
  } else {
    unsigned int i = blockIdx.x * blockSize + threadIdx.x;
    while (i < n) {
      mySum += g_idata[i];
      i += gridSize;
    }
  }

  // Reduce within warp using shuffle or reduce_add if T==int & CUDA_ARCH ==
  // SM 8.0
  mySum = warpReduceSum(mask, mySum);

  // each thread puts its local sum into shared memory
  if ((tid % warpSize) == 0) {
    sdata[tid / warpSize] = mySum;
  }

  __syncthreads();

  const unsigned int shmem_extent =
      (blockSize / warpSize) > 0 ? (blockSize / warpSize) : 1;
  const unsigned int ballot_result = __ballot_sync(mask, tid < shmem_extent);
  if (tid < shmem_extent) {
    mySum = sdata[tid];
    // Reduce final warp using shuffle or reduce_add if T==int & CUDA_ARCH ==
    // SM 8.0
    mySum = warpReduceSum(ballot_result, mySum);
  }

  // write result for this block to global mem
  if (tid == 0) {
    g_odata[blockIdx.x] = mySum;
  }
}

extern "C" void reduceSeven(double *g_idata, double *g_odata,
                        unsigned int n){
                            int smemSize = ((blockSize / 32) + 1) * sizeof(int);
                            int gridSize = n / blockSize;
                            reduce7<<<gridSize,blockSize,smemSize>>>(g_idata, g_odata, n);
                        }

extern "C" void initialize_CUDA(int rank){
    int cudaDeviceCount;
    int cE = hipGetDeviceCount( &cudaDeviceCount);
    if( cE != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n",
            cE, cudaDeviceCount );
        exit(-1);
    }
    cE = hipGetDeviceCount( &cudaDeviceCount);
    if( cE != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
            rank, (rank % cudaDeviceCount), cE);
        exit(-1);
    }
}